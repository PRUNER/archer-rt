#include "swordrt_common.h"
#include <stdio.h>

#define CUDA_WARN(XXX) \
    do { if (XXX != hipSuccess) printf("CUDA Error: %s[%d], at line %d\n", \
    hipGetErrorString(XXX), XXX, __LINE__); hipDeviceSynchronize(); } while (0)


AccessInfo *host_accesses[THREAD_NUM];
AccessInfo *device_accesses;
hipStream_t stream[THREAD_NUM];

extern thread_local int tid;

void malloc_host() {
	CUDA_WARN(hipHostMalloc((void**)&host_accesses[tid], sizeof(AccessInfo) * NUM_OF_ACCESSES));
}

void malloc_device() {
	// printf("Bytes: %lu\n", sizeof(AccessInfo) * TOTAL_ACCESSES * THREAD_NUM);
	CUDA_WARN(hipMalloc((void**) &device_accesses, sizeof(AccessInfo) * TOTAL_ACCESSES * THREAD_NUM));
	CUDA_WARN(hipStreamCreate(&stream[tid]));
}

void set_device() {
	CUDA_WARN(hipSetDevice(3));
}

void copy_to_device(int chunk) {
	// hipError_t res = hipMemcpy(device_accesses + ((sizeof(AccessInfo) * NUM_OF_ACCESSES) * chunk) + (tid * TOTAL_ACCESSES),
	//  		   host_accesses[tid], sizeof(AccessInfo) * NUM_OF_ACCESSES, hipMemcpyHostToDevice);
	uint64_t offset = ((sizeof(AccessInfo) * NUM_OF_ACCESSES) * chunk) + (tid * TOTAL_ACCESSES);
	CUDA_WARN(hipMemcpy(device_accesses,
			   host_accesses[tid],
			   sizeof(AccessInfo) * NUM_OF_ACCESSES,
			   hipMemcpyHostToDevice));
			   /*
	CUDA_WARN(hipMemcpyAsync(device_accesses,
			   host_accesses[tid],
			   sizeof(AccessInfo) * NUM_OF_ACCESSES,
			   hipMemcpyHostToDevice,
			   stream[tid]));
			   */
}

